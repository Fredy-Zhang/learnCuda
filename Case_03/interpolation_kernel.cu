#include "hip/hip_runtime.h"
#include <torch/extension.h>

/*
__host__ : call CPU and execute in CPU
__device__: call GPU and execute in GPU

__global__: call in CPU and execute in GPU.
*/

/*
In cuda coding: only have the void type for function.
*/

template <typename scalar_t>
__global__ void trilinear_fw_kernel (
    const torch::PackedTensorAccessor<scalar_t, 3, torch::RestrictTraits, size_t> feats,
    const torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictTraits, size_t> points,
    torch::PackedTensorAccessor<scalar_t, 2, torch::RestrictTraits, size_t> feat_interp
){
    /*
      1. Getting the thread ID;
      2. filter the useless threads.
    */
    // getting the thread ID.
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    const int f = blockIdx.y * blockDim.y + threadIdx.y;

    if (n >= feats.size(0) || f >= feats.size(2)) return;
    
    // point (-1~1), need to normalize
    const scalar_t u = (points[n][0]+1)/2;
    const scalar_t v = (points[n][1]+1)/2;
    const scalar_t w = (points[n][2]+1)/2;

    const scalar_t a = (1-v)*(1-w);
    const scalar_t b = (1-v)*w;
    const scalar_t c = v*(1-w);
    const scalar_t d = 1-a-b-c;
    feat_interp[n][f] = (1-u)*(a*feats[n][0][f] + 
                                b*feats[n][1][f] +
                                c*feats[n][2][f] +
                                d*feats[n][3][f]) + 
                        u*(a*feats[n][4][f] + 
                            b*feats[n][5][f] +
                            c*feats[n][6][f] + 
                            d*feats[n][7][f]);
}






// fw: forward: put input to get the output.
// bw: backward: output to update the parameters.
// cu: cuda
torch::Tensor trilinear_fw_cu(
    torch::Tensor feats,
    torch::Tensor point
){
    // generate the output tensor
    const int N = feats.size[0], F = feat.size(2);

    // should put into same cuda device.
    torch::Tensor feat_interp = torch::zeros({N, F}, feats.options());
    
    // set the data type of variable, and put this variable into same device
    //  torch::zeeros({N, F}, torch::dtype(torch::kInt32).device(feats.device));
    
    // every thread operates one point calculates. 

    // At first, defining the number of Blocks in Grids, and need how many threads.
    // 1. the N points can parallel computes;
    // 2. the F features can parallel computes;

    // two parts can be paralleled.
    // taking the two threads. 16*16 = 256
    const dim3 threads(16, 16); // max thread 256
    const dim3 blocks((N+threads.x-1)/threads.x, (F+threads.y-1)/threads.y);

    // throw to kernel.
    // AT_DISPATCH_FLOATING_TYPES_HALF 16bits calculated

    /* 
    scalar_t: place_holder <if don't know the exact data type.>
    <<<blocks, threads>>>: number of blocks and number of threads
    Input and Ouputs: 
            feats.packed_accessor (packed_accessor: covert the tensor type to others)
            points.packed_accessor
            feat_interp.packed_accessor
        because cuda do not return the result.
    <scalar_t, 3, torch::RestrictTraits, size_t>: 
        <data_type of input, dimension of input, Individual storage, shape>
    */
    AT_DISPATCH_FLOATING_TYPES(feats.type(), "trilinear_fw_cu", 
    ([&] {
        // if adding a Int variable, just add in it, no need to packed_accessor
        // a,
        trilinear_fw_cu<scalar_t><<<blocks, threads>>>(
        feats.packed_accessor<scalar_t, 3, torch::RestrictTraits, size_t>(),
        points.packed_accessor<scalar_t, 2, torch::RestrictTraits, size_t>()
        feat_interp.packed_accessor<scalar_t, 2, torch::RestrictTraits, size_t>(),
        
        /* If already know the data type
        trilinear_fw_cu<<<blocks, threads>>>(
        feats.packed_accessor<float, 3, torch::RestrictTraits, size_t>(),
        points.packed_accessor<float, 2, torch::RestrictTraits, size_t>()
        feat_interp.packed_accessor<float, 2, torch::RestrictTraits, size_t>(),

        */
        
        // gates.data<scalar_t>(),
        // old_cell.data<scalar_t>(),
        // new_h.data<scalar_t>(),
        // new_cell.data<scalar_t>(),
        // input_gate.data<scalar_t>(),
        // output_gate.data<scalar_t>(),
        // candidate_cell.data<scalar_t>(),
        // state_size);
  }));

    // // if only one part need parallel.
    // const int threads = 256;
    // const dim3 threads(256);



}