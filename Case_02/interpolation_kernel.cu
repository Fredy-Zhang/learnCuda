#include "hip/hip_runtime.h"
#include <torch/extension.h>

// fw: forward: put input to get the output.
// bw: backward: output to update the parameters.
// cu: cuda
torch::Tensor trilinear_fw_cu(
    torch::Tensor feats,
    torch::Tensor point
){
    // generate the output tensor
    const int N = feats.size[0], F = feat.size(2);

    // should put into same cuda device.
    torch::Tensor feat_interp = torch::zeros({N, F}, feats.options());
    
    // set the data type of variable, and put this variable into same device
    //  torch::zeeros({N, F}, torch::dtype(torch::kInt32).device(feats.device));
    
    // every thread operates one point calculates. 

    // At first, defining the number of Blocks in Grids, and need how many threads.
    // 1. the N points can parallel computes;
    // 2. the F features can parallel computes;

    // two parts can be paralleled.
    // taking the two threads. 16*16 = 256
    const dim3 threads(16, 16); // max thread 256
    const dim3 blocks((N+threads.x-1)/threads.x, (F+threads.y-1)/threads.y);

    // throw to kernel.
    // AT_DISPATCH_FLOATING_TYPES_HALF 16bits calculated

    AT_DISPATCH_FLOATING_TYPES(feats.type(), "trilinear_fw_cu", 
    ([&] {
        trilinear_fw_cu<scalar_t><<<blocks, threads>>>(
        feats.packed_accessor<scalar_t, 3, torch::RestrictTraits, size_t>(),
        points.packed_accessor<scalar_t, 2, torch::RestrictTraits, size_t>()
        feat_interp.packed_accessor<scalar_t, 2, torch::RestrictTraits, size_t>(),
        // gates.data<scalar_t>(),
        // old_cell.data<scalar_t>(),
        // new_h.data<scalar_t>(),
        // new_cell.data<scalar_t>(),
        // input_gate.data<scalar_t>(),
        // output_gate.data<scalar_t>(),
        // candidate_cell.data<scalar_t>(),
        // state_size);
  }));

    // // if only one part need parallel.
    // const int threads = 256;
    // const dim3 threads(256);



}